
#include <hip/hip_runtime.h>
#include <stdio.h>

// __global__ is a C language construct called a "Declaration Specifier" or "DecelSpec" 
// this tells CUDA that this is a kernel not CPU code
__global__ void square(float *d_out, float *d_in)
{
	int idx = threadIdx.x; // get index of the thread
	float f = d_in[idx];
	d_out[idx] = f * f;
}

int main(int argc, char** argv)
{
	const int ARRAY_SIZE = 64;
	const int ARRAY_BYTES = ARRAY_SIZE * sizeof(float);

	// generate input array on host
	float h_in[ARRAY_SIZE];
	for (int i = 0; i < ARRAY_SIZE; ++i)
	{
		h_in[i] = float(i);
	}
	float h_out[ARRAY_SIZE];

	// declare GPU memory pointers
	float * d_in;
	float * d_out;

	// allocate GPU memory
	hipMalloc((void **) &d_in, ARRAY_BYTES);
	hipMalloc((void **) &d_out, ARRAY_BYTES);

	// copy array from host to device
	hipMemcpy(d_in, h_in, ARRAY_BYTES, hipMemcpyHostToDevice);

	// launch kernel
	square<<<1, ARRAY_SIZE>>>(d_out, d_in);

	// copy array back from device to host
	hipMemcpy(h_out, d_out, ARRAY_BYTES, hipMemcpyDeviceToHost);

	// print results
	for(int i = 0; i < ARRAY_SIZE; ++i)
	{
		printf("%f", h_out[i]);
		printf(((i%4)!=3)?"\t":"\n");
	}

	// free GPU memory
	hipFree(d_in);
	hipFree(d_out);

	return 0;
}
